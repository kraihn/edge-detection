#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <cutil.h>

/* 
 * In CUDA it is necessary to define block sizes
 * The grid of data that will be worked on is divided into blocks
 */
#define BLOCK_SIZE 4
/**
 * This is the function that will be executed on the stream processors
 * The __global__ directive identifies this function as being
 * an executable kernel on the CUDA device.
 * All kernesl must be declared with a return type void 
 */ 
__global__ void cu_img_process(int *img, int *edges, int height, int width, int threshold, float *filter){
    int i;
	int j;
	int pos;
	
	/* blockIdx.x is a built-in variable in CUDA
           that returns the blockId in the x axis.
	   blockIdx.y is a built-in variable in CUDA
           that returns the blockId in the y axis
       threadIdx.x is another built-in variable in CUDA
           that returns the threadId in the x axis
           of the thread that is being executed by this
           stream processor in this particular block
       threadIdx.y is another built-in variable in CUDA
           that returns the threadId in the y axis
           of the thread that is being executed by this
           stream processor in this particular block
        */
	i=blockIdx.x*BLOCK_SIZE+threadIdx.x;
	j=blockIdx.y*BLOCK_SIZE+threadIdx.y;
	pos = i*height+ j;
		
	float sum = 0.0;        
	for(int g = 0; g < 3; g++){
		for(int h = 0; h < 3; h++){                    
			
			if((i == 0 && g == 0) || (i == height-1 && g == 2) || (j == 0 && h == 0) || (j == width-1 && h == 2))
			   continue;                    
			
			sum += filter[g*3+h] * img[(i-1+g)*height + (j-1+h)];
		}
	}
	sum = round(sum);
	if(sum > threshold)
		edges[pos] = 255;
	else
		edges[pos] = 0;
}

extern "C++" void img_process(int *img, int *edges, int height, int width, int threshold){
	//block_d and thread_d are the GPU counterparts of the arrays that exist on the host memory 
	int *img_d;
	int *edges_d;
	float *filter_d;
	
	float filter[] = {-0.0625,-0.0625,-0.0625,-0.0625,0.5,-0.0625,-0.0625,-0.0625,-0.0625};

	//int nBlocks;
	hipError_t result;
	
	//allocate memory on device
	// hipMalloc allocates space in the memory of the GPU
	result = hipMalloc((void**)&img_d,sizeof(int)*height*width);
	if (result != hipSuccess) {
		printf("hipMalloc - img_d - failed\n");
		exit(1);
	}
	result = hipMalloc((void**)&edges_d,sizeof(int)*height*width);
	if (result != hipSuccess) {
		printf("hipMalloc - edges_d - failed\n");
		exit(1);
	}
	result = hipMalloc((void**)&filter_d,sizeof(int)*3*3);
	if (result != hipSuccess) {
		printf("hipMalloc - filter_d - failed\n");
		exit(1);
	}
	
	//copy the array into the variable array_d in the device 
	result = hipMemcpy(img_d,img ,sizeof(int)*height*width,hipMemcpyHostToDevice);
	if (result != hipSuccess) {
		printf("hipMemcpy failed.");
		exit(1);
	}
	result = hipMemcpy(filter_d, filter,sizeof(int)*3*3,hipMemcpyHostToDevice);
	if (result != hipSuccess) {
		printf("hipMemcpy failed.");
		exit(1);
	}

	//execution configuration...
	// Indicate the dimension of the block
	dim3 dimblock(BLOCK_SIZE,BLOCK_SIZE);
	// Indicate the dimension of the grid 
	//nBlocks = arraySize/BLOCK_SIZE;
	dim3 dimgrid(height/BLOCK_SIZE, width/BLOCK_SIZE);
	//actual computation: Call the kernel
	cu_img_process<<<dimgrid,dimblock>>>(img_d, edges_d, height, width, threshold, filter_d);
	
	//read results back:
	// Copy the results from the memory in the GPU back to the memory on the host
	result = hipMemcpy(edges,edges_d,sizeof(int)*height*width,hipMemcpyDeviceToHost);
	if (result != hipSuccess) {
		printf("hipMemcpy - GPU to host - edges_d - failed\n");
		exit(1);
	}

	// Release the memory on the GPU 
	hipFree(img_d);
	hipFree(edges_d);
	hipFree(filter_d);
}